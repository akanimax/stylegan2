
#include <hip/hip_runtime.h>
#include <cstdio>

void checkCudaError(hipError_t err)
{
    if (err != hipSuccess)
    {
        printf("%s: %s\n", hipGetErrorName(err), hipGetErrorString(err));
        exit(1);
    }
}

__global__ void cudaKernel(void)
{
    printf("GPU says hello.\n");
}

int main(void)
{
    printf("CPU says hello.\n");
    checkCudaError(hipLaunchKernel(reinterpret_cast<const void*>((void*)cudaKernel), 1, 1, NULL, 0, NULL));
    checkCudaError(hipDeviceSynchronize());
    return 0;
}
